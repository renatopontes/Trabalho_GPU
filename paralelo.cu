#include "hip/hip_runtime.h"
#include "include/paralelo.h"

// Funções auxiliares --------------------------------------------------------------------

__device__ int pos_valida(int i, int j, GLOBALS *g) {
	return (i >= 0) && (i < g->n1) && (j >= 0) && (j < g->n2);
}

__device__ float get_v(float *malha, int i, int j, GLOBALS *g) {
	if (i < 0) return g->uo;
	if (i == g->n1) return g->ue;
	if (j < 0) return g->us;
	if (j == g->n2) return g->un;

	return malha[i*g->n2 + j];
}

void collect_globals(GLOBALS *g) {
	g->n1 = n1; g->n2 = n2;
	g->h1 = h1; g->h2 = h2;
	g->un = un; g->ue = ue; g->us = us; g->uo = uo;
	g->w_fixo = w_fixo;
	g->pi = pi;
}

// Sobre-relaxação sucessiva -------------------------------------------------------------

__global__ void processa_malha_w(float *malha, const int paridade, GLOBALS *g) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (pos_valida(i, j, g) && ((i+j) % 2 == paridade)) {
    	int n2 = g->n2;
		float o, e, s, n;
		float h1 = g->h1, h2 = g->h2;
		float w_fixo = g->w_fixo;
		float xi = (i+1.0)*g->h1;
		float yj = (j+1.0)*g->h2;

		o = (2.0 + h1 * get_a(xi, yj)) / (4.0 * (1.0 + sq(h1)/sq(h2)));
		e = (2.0 - h1 * get_a(xi, yj)) / (4.0 * (1.0 + sq(h1)/sq(h2)));
		s = (2.0 + h2 * get_b(xi, yj)) / (4.0 * (1.0 + sq(h2)/sq(h1)));
		n = (2.0 - h2 * get_b(xi, yj)) / (4.0 * (1.0 + sq(h2)/sq(h1)));

		malha[i*n2 + j] = (1 - w_fixo) * malha[i*n2 + j] + w_fixo *
			(o*get_v(malha,i-1,j,g) + e*get_v(malha,i+1,j,g) +
			s*get_v(malha,i,j-1,g) + n*get_v(malha,i,j+1,g));
    }
}

// Sobre-relaxação sucessiva local -------------------------------------------------------

__global__ void processa_malha_l(float *malha, const int paridade, GLOBALS *g) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (pos_valida(i, j, g) && ((i+j) % 2 == paridade)) {
    	int n2 = g->n2;
		float o, e, s, n, q, w_local;
		float h1 = g->h1, h2 = g->h2;
		float xi = (i+1.0)*g->h1;
		float yj = (j+1.0)*g->h2;
		float pi = g->pi;

		o = (2.0 + h1 * get_a(xi, yj)) / (4.0 * (1.0 + sq(h1)/sq(h2)));
		e = (2.0 - h1 * get_a(xi, yj)) / (4.0 * (1.0 + sq(h1)/sq(h2)));
		s = (2.0 + h2 * get_b(xi, yj)) / (4.0 * (1.0 + sq(h2)/sq(h1)));
		n = (2.0 - h2 * get_b(xi, yj)) / (4.0 * (1.0 + sq(h2)/sq(h1)));

		q = 2.0 * (sqrt(e*o) * cos(h1*pi) + sqrt(s*n) * cos(h2 * pi));
		w_local = 2.0 / (1 + sqrt(1 - sq(q)));

		malha[i*n2 + j] = (1 - w_local) * malha[i*n2 + j] + w_local *
			(o*get_v(malha,i-1,j,g) + e*get_v(malha,i+1,j,g) +
			s*get_v(malha,i,j-1,g) + n*get_v(malha,i,j+1,g));
    }
}

// Código para alocar e rodar os kernels -------------------------------------------------

// Método de Gauss-Seidel com sobre-relaxação sucessiva. w variável se modo == LOCAL.
void gauss_seidel_par(int iter, int modo) {
	float *malha_dev;
	GLOBALS *gd, gh;

	int n_bytes = n1 * n2 * sizeof(float);

	collect_globals(&gh);

	CUDA_SAFE_CALL(hipMalloc((void**) &malha_dev, n_bytes));
	CUDA_SAFE_CALL(hipMemcpy(malha_dev, malha, n_bytes, hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(hipMalloc((void**) &gd, sizeof(GLOBALS)));
	CUDA_SAFE_CALL(hipMemcpy(gd, &gh, sizeof(GLOBALS), hipMemcpyHostToDevice));
	
	dim3 n_threads(TAM_BLOCO, TAM_BLOCO);
	int a, b;
	a = n1/n_threads.x;
	a += n1 % n_threads.x ? 1 : 0;
	b = n2/n_threads.y;
	b += n2 % n_threads.y ? 1 : 0;
    dim3 blocos_grade(a, b);

    if (modo == FIXO) {
		while(iter--) {
			processa_malha_w<<<blocos_grade, n_threads>>>(malha_dev, PAR, gd);
			CUDA_SAFE_CALL(hipGetLastError());
			processa_malha_w<<<blocos_grade, n_threads>>>(malha_dev, IMPAR, gd);
			CUDA_SAFE_CALL(hipGetLastError());
		}
	} else {
		while(iter--) {
			processa_malha_l<<<blocos_grade, n_threads>>>(malha_dev, PAR, gd);
			CUDA_SAFE_CALL(hipGetLastError());
			processa_malha_l<<<blocos_grade, n_threads>>>(malha_dev, IMPAR, gd);
			CUDA_SAFE_CALL(hipGetLastError());
		}
	}


	CUDA_SAFE_CALL(hipMemcpy(malha, malha_dev, n_bytes, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(malha_dev));
	CUDA_SAFE_CALL(hipFree(gd));
	CUDA_SAFE_CALL(hipDeviceReset());
}