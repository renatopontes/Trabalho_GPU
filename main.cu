#include "hip/hip_runtime.h"
/* Alunos: 																				*/
/*			Renato Pontes Rodrigues														*/
/*			Mateus Ildefonso do Nascimento												*/

/* Para compilar:																		*/
/* 			make [TAM_BLOCO=T]															*/
/* 			T é um inteiro, o tamanho do bloco.	(default: 8)							*/

#include "include/global.h"
#include "include/sequencial.h"
#include "include/paralelo.h"

int n1, n2;
float un = 5, ue = 10, us = 5, uo = 0;
float h1, h2;
float *malha;
const float w_fixo = 1.3;
const float pi = 2.0*acos(0);

__host__ __device__ float get_a(float x, float y) {
	return 500.0 * x * (1.0 - x) * (0.5 - y);
}

__host__ __device__ float get_b(float x, float y) {
	return 500.0 * y * (1.0 - y) * (x - 0.5);
}

void init_malha() {
	srand(time(NULL));
	for (int i = 0; i < n1*n2; ++i) {
		malha[i] = (1.0 * rand() / RAND_MAX) * 10.0;
	}
}

int main(int argc, char **argv) {
	FILE *fout;
	TEMPO t;
	int iter;

	if (argc < 4) {
		SHOW_ERR("Passagem incorreta de parametros.\n\n"
			"\tUso: ./gauss_seidel N1 N2 iter [sw|sl|pw|pl]\n"
			"\tN1: largura da malha\n"
			"\tN2: altura da malha\n"
			"\titer: numero de iteracoes\n"
			"\tsw: processamento sequencial com sobre-relaxacao sucessiva. (default)\n"
			"\tsl: processamento sequencial com sobre-relaxacao sucessiva local.\n"
			"\tpw: processamento paralelo com sobre-relaxacao sucessiva.\n"
			"\tpl: processamento paralelo com sobre-relaxacao sucessiva local.\n");
	}

	n1 = atoi(argv[1]);
	n2 = atoi(argv[2]);
	iter = atoi(argv[3]);

	h1 = 1.0 / (n1 + 1);
	h2 = 1.0 / (n2 + 1);

	fout = fopen("out/matriz.txt", "w+");
	if (!fout) {
		SHOW_ERR("Nao foi possivel criar arquivo de saida\n");
	}

	malha = (float *) malloc(n1 * n2 * sizeof(float));

	init_malha();

	if (argc == 4 || !strcmp(argv[4], "sw")) {
		// printf("Processamento sequencial\n"
		// 	"Sobre-relaxacao sucessiva\n");
		t = gauss_seidel_seq(iter);
	}
	else if (!strcmp(argv[4], "sl")) {
		// printf("Processamento sequencial\n"
		// 	"Sobre-relaxacao sucessiva local\n");
		t = gauss_seidel_seq(iter, LOCAL);
	}
	else if (!strcmp(argv[4], "pw")) {
		// printf("Processamento paralelo\n"
		// 	"Sobre-relaxacao sucessiva\n");
		t = gauss_seidel_par(iter);
	}
	else if (!strcmp(argv[4], "pl")) {
		// printf("Processamento paralelo\n"
		// 	"Sobre-relaxacao sucessiva\n");
		t = gauss_seidel_par(iter, LOCAL);
	}

	for(int j = n2; j >= -1; --j) {
		for (int i = -1; i <= n1 ; ++i) {
			fprintf(fout, "%f ", get_v(i, j));
		}
		fprintf(fout, "\n");
	}

	printf("%.6f\t%.6f\t%.6f\t%.6f\n", t.ida, t.principal, t.volta, t.total);

	free(malha);

	return 0;
}