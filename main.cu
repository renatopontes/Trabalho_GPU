#include "hip/hip_runtime.h"
/* Alunos: 																				*/
/*			Renato Pontes Rodrigues														*/
/*			Mateus Ildefonso do Nascimento												*/

/* Para compilar:																		*/
/* 			make																		*/

#include "include/global.h"
#include "include/clock_timer.h"
#include "include/sequencial.h"
#include "include/paralelo.h"

int n1, n2;
float un = 5, ue = 10, us = 5, uo = 0;
float h1;
float h2;
float *malha;
const float w_fixo = 0.6;
const float PI = 2.0*acos(0);

COORD_MALHA valor(int i, int j) {
	float xi, yj;

	xi = (i+1)*h1;
	yj = (j+1)*h2;

	return COORD_MALHA(xi, yj);
}

float get_a(float x, float y) {
	return 500.0 * x * (1.0 - x) * (0.5 - y);
}

float get_b(float x, float y) {
	return 500.0 * y * (1.0 - y) * (x - 0.5);
}

float get_v(int i, int j) {
	if (i < 0) return uo;
	if (i == n1) return ue;
	if (j < 0) return us;
	if (j == n2) return un;

	return malha[i*n2 + j];
}

void init_malha() {
	srand(time(NULL));
	for (int i = 0; i < n1*n2; ++i) {
		malha[i] = (1.0 * rand() / RAND_MAX) * 10.0;
	}
}

int main(int argc, char **argv) {
	FILE *fout;
	const int iter = 1000;

	if (argc < 3) {
		SHOW_ERR("Passagem incorreta de parametros.\n\n"
			"\tUso: ./gauss_seidel N1 N2 [sw|sl|pw|pl]\n"
			"\tN1: largura da malha\n"
			"\tN2: altura da malha\n"
			"\tsw: processamento sequencial com sobre-relaxacao sucessiva. (default)\n"
			"\tsl: processamento sequencial com sobre-relaxacao sucessiva local.\n"
			"\tpw: processamento paralelo com sobre-relaxacao sucessiva.\n"
			"\tpl: processamento paralelo com sobre-relaxacao sucessiva local.\n");
	}

	n1 = atoi(argv[1]);
	n2 = atoi(argv[2]);

	h1 = 1.0 / (n1 + 1);
	h2 = 1.0 / (n2 + 1);

	fout = fopen("out/matriz.txt", "w+");
	if (!fout) {
		SHOW_ERR("Nao foi possivel criar arquivo de saida\n");
	}

	malha = (float *) malloc(n1 * n2 * sizeof(float));

	init_malha();

	if (argc == 3 || !strcmp(argv[3], "sw")){
		printf("Processamento sequencial\n"
			"Sobre-relaxacao sucessiva\n");
		gauss_seidel_seq_w(iter);
	}
	else if (!strcmp(argv[3], "sl")){
		printf("Processamento sequencial\n"
			"Sobre-relaxacao sucessiva local\n");
		gauss_seidel_seq_l(iter);
	}

	for(int j = n2; j >= -1; --j) {
		for (int i = -1; i <= n1 ; ++i) {
			fprintf(fout, "%f ", get_v(i, j));
		}
		fprintf(fout, "\n");
	}

	return 0;
}